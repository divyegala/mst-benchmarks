/***********************************************************************************
Implementing Minimum Spanning Tree on CUDA using Atomic Functions. Part of 
implementation done for the paper:

"Large Graph Algorithms for Massively Multithreaded Architectures"
Pawan Harish, Vibhav Vineet and P.J.Narayanan.
Technical Report IIIT/TR/2009/74, 
International Institute of Information Technology-Hyderabad

Copyright (c) 2009 International Institute of Information Technology - Hyderabad. 
All rights reserved.
  
Permission to use, copy, modify and distribute this software and its documentation for 
educational purpose is hereby granted without fee, provided that the above copyright 
notice and this permission notice appear in all copies of this software and that you do 
not sell the software.
  
THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
OTHERWISE.

Created by Pawan Harish and Vibhav Vineet
************************************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
using namespace std::chrono;

//#include "CUDAMST.h"
#include "CUDAMST.cu"

//void MSTGraph(int argc, char** argv);

////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
	no_of_nodes=0;
	source = 0 ;
	edge_list_size=0;
	MSTGraph( argc, argv);
	exit(0);
}



////////////////////////////////////////////////////////////////////////////////
//Apply MST on a Graph using CUDA
////////////////////////////////////////////////////////////////////////////////
void MSTGraph( int argc, char** argv) 
{
	printf("Reading File\n");
	fp = fopen(argv[1],"r");
	if(!fp)
	{
		printf("Error Reading graph file\n");
		return;
	}

	fscanf(fp,"%d",&no_of_nodes);

	hostMemAllocationNodes();
	// allocate host memory

	printf("Reading %d nodes	",no_of_nodes);
	// initalize the memory
	for(int i = 0; i < no_of_nodes; i++) 
	{
		fscanf(fp,"%d %d",&start,&edgeno);
		h_graph_nodes[i].starting = start;
		h_graph_nodes[i].no_of_edges = edgeno;
		sameindex[i]=i;
		falseval[i]=false;
		trueval[i]=true;
		infinity[i] = INF;
		zero[i]=0;
		h_maxid_maxdegree[i]=-1;

	}

	//read the source node from the file, not needed here though
	fscanf(fp,"%d",&source);

	fscanf(fp,"%d",&edge_list_size);
	printf("Reading %d edges\n",edge_list_size);

	int id,cost;
	hostMemAllocationEdges();

	for(int i=0; i < edge_list_size ; i++)
	{
		fscanf(fp,"%d",&id);
		fscanf(fp,"%d",&cost);
		h_graph_edges[i] = id;
		h_graph_weights[i] = cost;
		h_graph_MST_edges[i] = false;

	}

	if(fp)
		fclose(fp);    

	printf("Finished Reading File\n");

	printf("Copying Everything to GPU memory\n");


	//Copy the Node list to device memory
	
	deviceMemAllocateNodes() ; 
	deviceMemAllocateEdges() ; 
	deviceMemCopy();

	hipDeviceSynchronize();
	auto start = high_resolution_clock::now();
	GPUMST(); 
	
	// hipMemcpy( test, d_graph_colorindex, sizeof(int)*no_of_nodes, hipMemcpyDeviceToHost);
	// for(int i=0;i<no_of_nodes;i++)
	//  {
	// 	 if(test[i]!=0)
	// 	 {
	// 		 printf("All Colors not 0, Error at %d\n",i);
	// 		 break;
	// 	 }
	
	//  }

	int q=0;
	int minimumCost = 0 ; 
	printf("\nFinal edges present in MST\n");
	hipMemcpy( h_graph_MST_edges, d_graph_MST_edges, sizeof(bool)*edge_list_size, hipMemcpyDeviceToHost) ;
	for(int i=0;i<int(edge_list_size);i++)
	{
		if(h_graph_MST_edges[i])
		{
			
			int edge = i;
			int edgeweight = h_graph_weights[edge];
			minimumCost += edgeweight; 
			q++;
		}
	}
	printf("No of edges in MST: %d, no of nodes: %d cost %d\n",q,no_of_nodes,minimumCost);
	hipDeviceSynchronize();
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
	std::cout << "Finished in: " << duration.count() << std::endl;
	
	freeMem();
		
}
